#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Example of integrating CUDA functions into an existing
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

#include <GL/glew.h>

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <iostream>

// CUDA runtime
//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h> 

#include "cutilStuff.h"

#include "SharedBuffer.h"

extern "C" struct hipGraphicsResource* fromCudaRegisterBufferForRead(GLuint a_buff)         
{ 
  struct hipGraphicsResource* resource = NULL;
  if( hipGraphicsGLRegisterBuffer(&resource, a_buff, cudaGraphicsMapFlagsReadOnly) != hipSuccess)
    return NULL;  
} 

extern "C" struct hipGraphicsResource* fromCudaRegisterBufferForWrite(GLuint a_buff)        
{ 
  struct hipGraphicsResource* resource = NULL;
  if( hipGraphicsGLRegisterBuffer(&resource, a_buff, cudaGraphicsMapFlagsWriteDiscard) != hipSuccess)
    return NULL; 
  return resource; 
} 

extern "C" struct hipGraphicsResource* fromCudaRegisterBufferForReadAndWrite(GLuint a_buff) 
{ 
  struct hipGraphicsResource* resource = NULL;
  if( hipGraphicsGLRegisterBuffer(&resource, a_buff, cudaGraphicsMapFlagsNone) != hipSuccess)
    return NULL; 
  return resource; 
} 

extern "C" void  fromCudaUnregisterBuffer(struct hipGraphicsResource* a_pResource) 
{
  if(a_pResource == NULL)
    return;

  CUDA_SAFE_CALL(hipGraphicsUnregisterResource(a_pResource));
}

extern "C" void* fromCudaMapRegisteredBuffer(struct hipGraphicsResource*& a_pGraphRes) 
{ 
  void* buffer = NULL;
  size_t numBytes = 0; 
  
  CUDA_SAFE_CALL(hipGraphicsMapResources(1, &a_pGraphRes, 0));
  CUDA_SAFE_CALL(hipGraphicsResourceGetMappedPointer((void**)&buffer, &numBytes, a_pGraphRes));

  return buffer; 
}

extern "C" void fromCudaUnmapRegisteredBuffer(struct hipGraphicsResource*& a_pGraphRes) 
{
  CUDA_SAFE_CALL(hipGraphicsUnmapResources(1, &a_pGraphRes, 0));
}



#define BLOCK_SIZE 512


__global__ void append(float* data, int* mask, int* append_index, float* res, int a_size)
{
	const int i(blockIdx.x * BLOCK_SIZE + threadIdx.x);
	if(i>=a_size)
    return;
  
  float val = data[i];
	if(mask[i])
		res[atomicAdd(append_index, 1)] = val;
}

#define NUM_OF_TESTS 10

void make_tests(int ARRAY_SIZE)
{
	srand(unsigned int(__TIME__));
	FILE* fd = fopen("Input.txt", "w");
	for(int j = 0; j < NUM_OF_TESTS; j++)
		for(int i = 0; i < ARRAY_SIZE; i++)
		{
			fprintf(fd, "%d ", rand() > RAND_MAX / 2 ? 1 : 0);
		}
	fclose(fd);
}

extern "C" int makeAppendWithCUDAKernelTest(int ARRAY_SIZE, bool createTests)
{
  float* host_array = new float[ARRAY_SIZE];
	int*   host_mask  = new   int[ARRAY_SIZE];
	

  if(createTests)
  {
    make_tests(ARRAY_SIZE);
    printf("Tests are ready\n");
	}


	FILE* fd = fopen("Input.txt", "r");
	if(fd == NULL)
	{
		printf("Couldn't open the input file\n");
		getchar();
		return 1;
	}

	for(int i = 0; i < ARRAY_SIZE; i++)
	{
		host_array[i] = float(i + 1);
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  //
  //
  SharedBuffer dataArray(host_array, ARRAY_SIZE * sizeof(float));
  SharedBuffer dataMask(host_mask, ARRAY_SIZE * sizeof(int));
  SharedBuffer dataResult(host_array, ARRAY_SIZE * sizeof(float));

  float* device_array  = (float*)dataArray.mapToCUDA();          // CUDA_CHECK_ERROR(hipMalloc((void**)&device_array,   ARRAY_SIZE * sizeof(float)));
  int*   device_mask   = (int*)dataMask.mapToCUDA();             // CUDA_CHECK_ERROR(hipMalloc((void**)&device_mask,    ARRAY_SIZE * sizeof(float)));
  float* device_result = (float*)dataResult.mapToCUDAForWrite(); // CUDA_CHECK_ERROR(hipMalloc((void**)&device_result,  ARRAY_SIZE * sizeof(float)));
  
  int* device_append_index;
	CUDA_CHECK_ERROR(hipMalloc((void**)&device_append_index, sizeof(int)));

	float times[NUM_OF_TESTS] = {0.0f};
	float average_time = 0.0f;

	printf("Array size: %d floats \nBlock size: %d threads\nMaking %d tests \n", ARRAY_SIZE, BLOCK_SIZE, NUM_OF_TESTS);

	for(int k = 0; k < NUM_OF_TESTS; k++)
	{
		for(int i = 0; i < ARRAY_SIZE; i++)
			fscanf(fd, "%d", &(host_mask[i]));

		CUDA_CHECK_ERROR(hipMemcpy(device_array, host_array, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK_ERROR(hipMemcpy(device_mask,  host_mask,  ARRAY_SIZE * sizeof(int),   hipMemcpyHostToDevice));

		CUDA_CHECK_ERROR(hipMemset(device_result,  0, ARRAY_SIZE * sizeof(float)));
		CUDA_CHECK_ERROR(hipMemset(device_append_index, 0, sizeof(int)));
	
		CUDA_CHECK_ERROR(hipEventRecord(start, 0));

		append<<<(ARRAY_SIZE / BLOCK_SIZE + 1), BLOCK_SIZE>>>(device_array, device_mask, device_append_index, device_result, ARRAY_SIZE);
		CUDA_CHECK_ERROR(hipGetLastError());

		CUDA_CHECK_ERROR(hipEventRecord(stop, 0));
		CUDA_CHECK_ERROR(hipEventSynchronize(stop));

		CUDA_CHECK_ERROR(hipEventElapsedTime(&(times[k]), start, stop));

		printf("Time spent executing the test %d by the GPU: %.8f milliseconds\n", k, times[k]);

		average_time += times[k];
	}
	average_time /= float(NUM_OF_TESTS);
	
	printf("Average time: %.8f milliseconds\n", average_time);
	getchar();

	fclose(fd);

  CUDA_CHECK_ERROR(hipFree(device_append_index));

	CUDA_CHECK_ERROR(hipEventDestroy(start));
	CUDA_CHECK_ERROR(hipEventDestroy(stop));

	delete[] host_array;
	delete[] host_mask;

  return 0;
}



